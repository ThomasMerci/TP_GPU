#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Template project which demonstrates the basics on how to setup a project 
 * example application.
 * Host code.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <iostream>
using namespace std;

// includes CUDA
#include <hip/hip_runtime.h>

#include "saxpy.h"

__global__ void saxpy_kernel(float *vector_SAXPY, float A, float *vector_X, float *vector_Y, int N)
{
	// A VOUS DE CODER

}



////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);




////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{

	runTest( argc, argv);
}

__host__ static int iDivUp(int a, int b) {
	return ((a % b != 0) ? (a / b + 1): (a/b));
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
	hipError_t error;

	unsigned long int N=256*1024;

	const unsigned int mem_size = N*sizeof(float);
	// allocate host memory
	float* h_vector_X = (float*) malloc(mem_size);
	float* h_vector_Y = (float*) malloc(mem_size);

	//Initilaisation des données d'entrée
	float A=1.0;

	for (int i=0;i<N;i++){
		h_vector_X[i]=(float)rand();
		h_vector_Y[i]=(float)rand();
	}


	////////////////////////////////////////////////////////////////////////////////
	// EXECUTION SUR LE CPU
	///////////////////////////////////////////////////////////////////////


	// Image trait�e sur le CPU
	float* h_vector_SAXPY_CPU = (float*) malloc( mem_size);

	printf("SAXPY CPU\n");

	hipEvent_t start,stop;
	error = hipEventCreate(&start);
	error = hipEventCreate(&stop);

	// Record the start event
	error = hipEventRecord(start, NULL);
	error = hipEventSynchronize(start);
	//Seuillage sur CPU
	// A VOUS DE CODER

	// Record the start event
	error = hipEventRecord(stop, NULL);
	// Wait for the stop event to complete
	error = hipEventSynchronize(stop);
	float msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);


	printf("CPU execution time %f\n",msecTotal);



	////////////////////////////////////////////////////////////////////////////////
	// EXECUTION SUR LE GPU
	///////////////////////////////////////////////////////////////////////

	printf("SAXPY GPU\n");

float* h_vector_SAXPY_GPU = (float*) malloc(mem_size);

	// images on device memory
	float* d_vector_X;
	float* d_vector_Y;
	float* d_vector_SAXPY;


	hipEvent_t start_mem,stop_mem;
	error = hipEventCreate(&start_mem);
	error = hipEventCreate(&stop_mem);

	error = hipEventRecord(start, NULL);
	error = hipEventSynchronize(start);



	
	// Alocation mémoire de d_vector_X, d_vector_Y et d_vector_SAXPY sur la carte GPU
	// A VOUS DE CODER

	// copy host memory to device
	// A VOUS DE CODER


	error = hipEventRecord(stop_mem, NULL);
	// Wait for the stop event to complete
	error = hipEventSynchronize(stop_mem);
	float msecMem = 0.0f;
	error = hipEventElapsedTime(&msecMem, start, stop_mem);

	// setup execution parameters -> découpage en threads
	// A VOUS DE CODER


	// lancement des threads executé sur la carte GPU
	// A VOUS DE CODER

	error = hipEventRecord(start_mem, NULL);
	error = hipEventSynchronize(start_mem);

	// copy result from device to host
	// A VOUS DE CODER

	// cleanup device memory
	// COMMENTAIRES A ENLEVER
	//hipFree(d_vector_X);
	//hipFree(d_vector_Y);
	//hipFree(d_vector_SAXPY);


	error = hipEventRecord(stop, NULL);
	// Wait for the stop event to complete
	error = hipEventSynchronize(stop);
	msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);
	float msecMem2 =0.0f;
	error = hipEventElapsedTime(&msecMem2, start_mem, stop);
	msecMem+=msecMem2;

	printf("GPU execution time %f ms (memory management %2.2f \%)\n",msecTotal,(msecMem)/(msecTotal)*100);

	float sum_diff=0;
for(int i=0;i<N;i++)
	sum_diff+= h_vector_SAXPY_GPU[i]-h_vector_SAXPY_CPU[i];

	printf("sum_diff = %f\n",sum_diff);

	// cleanup memory
	free(h_vector_X);
	free(h_vector_Y);
	free(h_vector_SAXPY_GPU);
	free(h_vector_SAXPY_CPU);
}
