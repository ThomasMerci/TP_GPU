#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/* Template project which demonstrates the basics on how to setup a project 
 * example application.
 * Host code.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <iostream>
using namespace std;

// includes CUDA
#include <hip/hip_runtime.h>

#include "seuillage.h"

#define TAILLE_BLOC_X 16
#define TAILLE_BLOC_Y 16

__global__ void seuillage_kernel(float d_image_in[][SIZE_J][SIZE_I],float d_image_out[][SIZE_J][SIZE_I])
{
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int j = blockDim.y * blockIdx.y + threadIdx.y;

	d_image_out[0][j][i] = d_image_in[0][j][i];
	((d_image_in[0][j][i]/sqrt(d_image_in[0][j][i]*d_image_in[0][j][i] + d_image_in[1][j][i]*d_image_in[1][j][i] + d_image_in[2][j][i]*d_image_in[2][j][i])) > 0.7)?(d_image_out[1][j][i] = d_image_in[0][j][i]):(d_image_out[1][j][i] = d_image_in[1][j][i]);
	d_image_out[2][j][i] = d_image_in[2][j][i];

}



////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);




////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{

	runTest( argc, argv);
}

__host__ static int iDivUp(int a, int b) {
	return ((a % b != 0) ? (a / b + 1): (a/b));
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
	hipError_t error;

	if (argc<2)
		printf("indiquer le chemin du repertoire contenant les images\n");

	const unsigned int mem_size = sizeof(float) * 3* SIZE_J * SIZE_I;
	// allocate host memory
	float* h_image_in = (float*) malloc(mem_size);


	//Initilaisation du volume d'entr�e
	FILE *file_ptr;
	char name_file_in[512];
	sprintf(name_file_in,"%s/ferrari.raw",argv[1]);
	printf("%s",name_file_in);
	file_ptr=fopen(name_file_in,"rb");
	fread(h_image_in,sizeof(float),3*SIZE_J*SIZE_I,file_ptr);
	fclose(file_ptr);


	////////////////////////////////////////////////////////////////////////////////
	// EXECUTION SUR LE CPU
	///////////////////////////////////////////////////////////////////////


	// Image trait�e sur le CPU
	float* h_image_out_CPU = (float*) malloc( mem_size);

	printf("Seuillage CPU d'une image couleur \n");

	hipEvent_t start,stop;
	error = hipEventCreate(&start);
	error = hipEventCreate(&stop);

	// Record the start event
	error = hipEventRecord(start, NULL);
	error = hipEventSynchronize(start);
	//Seuillage sur CPU
	seuillage_C( (float (*)[SIZE_J][SIZE_I])h_image_out_CPU, (float (*)[SIZE_J][SIZE_I])h_image_in);

	// Record the start event
	error = hipEventRecord(stop, NULL);
	// Wait for the stop event to complete
	error = hipEventSynchronize(stop);
	float msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);


	printf("CPU execution time %f\n",msecTotal);

	//Sauvegarde de l'image resultat
	char name_file_out_CPU[512];
	sprintf(name_file_out_CPU,"%s/ferrari_out_CPU.raw",argv[1]);
	file_ptr=fopen(name_file_out_CPU,"wb");
	fwrite(h_image_out_CPU,sizeof(float),3*SIZE_J*SIZE_I,file_ptr);
	fclose(file_ptr);


	////////////////////////////////////////////////////////////////////////////////
	// EXECUTION SUR LE GPU
	///////////////////////////////////////////////////////////////////////

	hipEvent_t start_mem,stop_mem;
	error = hipEventCreate(&start_mem);
	error = hipEventCreate(&stop_mem);

	error = hipEventRecord(start, NULL);
	error = hipEventSynchronize(start);


	float* h_image_out_GPU = (float*) malloc(mem_size);

	// images on device memory
	float* d_image_in;
	float* d_image_out;

	unsigned long taille_totale = 3*SIZE_J*SIZE_I*sizeof(float);

	// Alocation mémoire de d_image_in et d_image_out sur la carte GPU
	hipMalloc( (void**)&d_image_in, taille_totale);
	hipMalloc( (void**)&d_image_out, taille_totale);

	// copy host memory to device
	hipMemcpy(d_image_in, h_image_in, taille_totale, hipMemcpyHostToDevice);

	error = hipEventRecord(stop_mem, NULL);
	// Wait for the stop event to complete
	error = hipEventSynchronize(stop_mem);
	float msecMem = 0.0f;
	error = hipEventElapsedTime(&msecMem, start, stop_mem);

	// setup execution parameters -> découpage en threads
	dim3 threadsParBloc(TAILLE_BLOC_X, TAILLE_BLOC_Y);
	dim3 tailleGrille(iDivUp(SIZE_I, TAILLE_BLOC_X), iDivUp(SIZE_J, TAILLE_BLOC_Y));



	seuillage_kernel<<<tailleGrille, threadsParBloc>>>((float (*)[SIZE_J][SIZE_I])d_image_in, (float (*)[SIZE_J][SIZE_I])d_image_out);

	// lancement des threads executé sur la carte GPU
	// A VOUS DE CODER
	// INDICATION : pour les parametres de la fonction kernel seuillage_kernel, vous ferez un changement de type (float *) vers  (float (*)[SIZE_J][SIZE_I])
	// inspirez vous du lancement de la fonction seuillage_C dans le main.


	error = hipEventRecord(start_mem, NULL);
	error = hipEventSynchronize(start_mem);

	// copy result from device to host
	hipMemcpy(h_image_out_GPU, d_image_out, taille_totale, hipMemcpyDeviceToHost);


	// cleanup device memory
	hipFree(d_image_in);
	hipFree(d_image_out);


	error = hipEventRecord(stop, NULL);
	// Wait for the stop event to complete
	error = hipEventSynchronize(stop);
	msecTotal = 0.0f;
	error = hipEventElapsedTime(&msecTotal, start, stop);
	float msecMem2 =0.0f;
	error = hipEventElapsedTime(&msecMem2, start_mem, stop);
	msecMem+=msecMem2;

	printf("GPU execution time %f ms (memory management %2.2f \%)\n",msecTotal,(msecMem)/(msecTotal)*100);

	// Enregistrement de l'image de sortie sur un fichier
	char name_file_out_GPU[512];
	sprintf(name_file_out_GPU,"%s/ferrari_out_GPU.raw",argv[1]);
	file_ptr=fopen(name_file_out_GPU,"wb");
	fwrite(h_image_out_GPU,sizeof(float),3*SIZE_J*SIZE_I,file_ptr);
	fclose(file_ptr);


	// cleanup memory
	free(h_image_in);
	free(h_image_out_GPU);
	free(h_image_out_CPU);
}
